#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include "smoothlife.hpp"
#include "smoothlife_cuda.cuh"

#define BLOCK_SIZE 16 // threadsPerBlock(16, 16)
#define HALO 12

__device__ void complex_vector_multiplication(hipfftDoubleComplex* signal, hipfftDoubleComplex* filter, hipfftDoubleComplex* result) {
    float signal_real = (*signal).x;
    float signal_imag = (*signal).y;

    float filter_real = (*filter).x;
    float filter_imag = (*filter).y;

    // Perform complex multiplication
    (*result).x = signal_real * filter_real - signal_imag * filter_imag;  // Real part
    (*result).y = signal_real * filter_imag + signal_imag * filter_real;  // Imaginary part
}


void SmoothLife::applyCudaUpdate() {
    hipDeviceReset();

    // take Hermitian symmetry of a real-valued signal into account AND individual block + halo dims
    size_t num_coeffs = grid_size * (grid_size / 2 + 1);
    size_t num_coeffs_filter = (BLOCK_SIZE + 2 * HALO) * (BLOCK_SIZE + 2 * HALO);

    hipfftDoubleReal* d_field_real;
    hipfftDoubleComplex* d_field_fft;
    hipfftDoubleComplex* d_field_fft_M;
    hipfftDoubleComplex* d_field_fft_N;
    
    hipfftDoubleReal* d_field_real_output;

    hipMalloc(&d_field_real, sizeof(hipfftDoubleReal) * grid_size * grid_size);
    hipMalloc(&d_field_fft, sizeof(hipfftDoubleComplex) * num_coeffs);

    hipMalloc(&d_field_fft_M, sizeof(hipfftDoubleComplex) * num_coeffs);
    hipMalloc(&d_field_fft_N, sizeof(hipfftDoubleComplex) * num_coeffs);
    hipMalloc(&d_field_real_output, sizeof(hipfftDoubleReal) * grid_size * grid_size);


    hipMemcpy(d_field_real, input_weights_real, sizeof(double) * grid_size * grid_size, hipMemcpyHostToDevice);

    hipfftHandle plan; 
     
    hipfftPlan2d(&plan, grid_size, grid_size, HIPFFT_D2Z);
    hipfftExecD2Z(plan, d_field_real, d_field_fft);

    std::cout << "ejoo123456" << std::endl;

    hipfftDoubleComplex* h_disk_coeffs = new hipfftDoubleComplex[num_coeffs_filter];
    hipfftDoubleComplex* h_annulus_coeffs = new hipfftDoubleComplex[num_coeffs_filter];

    for (int i = 0; i < num_coeffs_filter; i++) {
        h_disk_coeffs[i].x = disk_weights_cuda[i][0];
        h_disk_coeffs[i].y = 0;
        h_annulus_coeffs[i].x = annulus_weights_cuda[i][0];
        h_annulus_coeffs[i].y = 0;
    }

    hipfftDoubleComplex* d_disk_coeffs;
    hipfftDoubleComplex* d_annulus_coeffs;
    hipMalloc(&d_disk_coeffs, sizeof(hipfftDoubleComplex) * num_coeffs);
    hipMalloc(&d_annulus_coeffs, sizeof(hipfftDoubleComplex) * num_coeffs);

    hipMemcpy(d_disk_coeffs, h_disk_coeffs, sizeof(hipfftDoubleComplex) * num_coeffs, hipMemcpyHostToDevice);
    hipMemcpy(d_annulus_coeffs, h_annulus_coeffs, sizeof(hipfftDoubleComplex) * num_coeffs, hipMemcpyHostToDevice);

    // No longer needed after the hipMemcpy
    delete[] h_disk_coeffs;
    delete[] h_annulus_coeffs;

    std::cout << "Izdrzao" << std::endl;

    int reduced_grid_size = grid_size / 2 + 1; // hermitian
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    int numBlocksX = (reduced_grid_size + threadsPerBlock.x - 1) / threadsPerBlock.x;
    int numBlocksY = (grid_size + threadsPerBlock.y - 1) / threadsPerBlock.y;
    dim3 numBlocks(numBlocksX, numBlocksY);
    // dim3 numBlocks((reduced_grid_size + threadsPerBlock.x - 1) / threadsPerBlock.x,
    //                (reduced_grid_size + threadsPerBlock.y - 1) / threadsPerBlock.y);
                   
    size_t height = grid_size;
    size_t width = reduced_grid_size;

    smooth_life_kernel<<<numBlocks, threadsPerBlock>>>(d_field_fft, d_disk_coeffs, d_annulus_coeffs, d_field_fft_M, d_field_fft_N, height, width);

    std::cout << "Jjss" << std::endl;
    
    hipfftDoubleComplex* h_field_fft_M = new hipfftDoubleComplex[num_coeffs];
    hipfftDoubleComplex* h_field_fft_N = new hipfftDoubleComplex[num_coeffs];
    hipMemcpy(h_field_fft_M, d_field_fft_M, sizeof(hipfftDoubleComplex) * num_coeffs, hipMemcpyDeviceToHost);
    hipMemcpy(h_field_fft_N, d_field_fft_N, sizeof(hipfftDoubleComplex) * num_coeffs, hipMemcpyDeviceToHost);
    for (size_t i = 0; i < num_coeffs; i++) {
        std::cout << h_field_fft_M[i].x;
        std::cout << h_field_fft_N[i].x;
    }
    // hipMemcpy(&field[0][0], d_field, sizeof(double) * grid_size * grid_size, hipMemcpyDeviceToHost);
    // hipFree(d_field);
}

// CUDA Kernel for SmoothLife with Shared Memory
__global__ void smooth_life_kernel(hipfftDoubleComplex* d_field_fft, hipfftDoubleComplex* d_disk_coeffs, hipfftDoubleComplex* d_annulus_coeffs, 
                                    hipfftDoubleComplex* d_field_fft_M, hipfftDoubleComplex* d_field_fft_N, size_t height, size_t width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;      // global
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = threadIdx.x + HALO;                        // local
    int ty = threadIdx.y + HALO;

    // Allocate shared memory for the block, including the halo region
    __shared__ hipfftDoubleComplex shared_grid[BLOCK_SIZE + 2 * HALO][BLOCK_SIZE + 2 * HALO];

    // Load cells and neighbors into shared memory
    if (x < width && y < height) {
        shared_grid[ty][tx] = d_field_fft[y * height + x];

        // Load the halo cells (neighbors)
        if (threadIdx.x < HALO) {
            shared_grid[ty][tx - HALO] = d_field_fft[y * height + ((x - HALO + width) % width)];
            shared_grid[ty][tx + BLOCK_SIZE] = d_field_fft[y * height + ((x + BLOCK_SIZE) % width)];
        }
        if (threadIdx.y < HALO) {
            shared_grid[ty - HALO][tx] = d_field_fft[((y - HALO + height) % height) * height + x];
            shared_grid[ty + BLOCK_SIZE][tx] = d_field_fft[((y + BLOCK_SIZE) % height) * height + x];
        }
        // Load corners of halo (optional but ensures full coverage)
        // if (threadIdx.x < HALO && threadIdx.y < HALO) {
        //     shared_grid[ty - HALO][tx - HALO] = d_field_fft[((y - HALO + height) % height) * height + ((x - HALO + width) % width)];
        //     shared_grid[ty - HALO][tx + BLOCK_SIZE] = d_field_fft[((y - HALO + height) % height) * height + ((x + BLOCK_SIZE) % width)];
        //     shared_grid[ty + BLOCK_SIZE][tx - HALO] = d_field_fft[((y + BLOCK_SIZE) % height) * height + ((x - HALO + width) % width)];
        //     shared_grid[ty + BLOCK_SIZE][tx + BLOCK_SIZE] = d_field_fft[((y + BLOCK_SIZE) % height) * height + ((x + BLOCK_SIZE) % width)];
        // }
    }
    __syncthreads();

    // Ensure we're within grid bounds and calculate M and N
    if (x < width && y < height) {
        
        hipfftDoubleComplex* shared_value = &(shared_grid[ty][tx]);
        hipfftDoubleComplex* filter_value_disk = &(d_disk_coeffs[y * (BLOCK_SIZE + 2 * HALO) + x]);
        hipfftDoubleComplex* filter_value_annulus = &(d_annulus_coeffs[y * (BLOCK_SIZE + 2 * HALO) + x]);
    
        // M_f
        complex_vector_multiplication(shared_value, filter_value_disk, d_field_fft_M);
        // N_f
        complex_vector_multiplication(shared_value, filter_value_annulus, d_field_fft_N);
    }
}
